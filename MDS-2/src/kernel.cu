#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include "kernel.hpp"
#include <cstdio>
using namespace std;


/*
 * calculate vec1 + vec2
 */
__global__ void kernel_vector_add(VALTUE_TYPE *X, VALTUE_TYPE *delta, int size, float alpha){
	unsigned int tid = threadIdx.x;
	unsigned int blockId = blockIdx.x;
	unsigned int offset = blockId * blockDim.x + tid;

	if(offset < size){
		X[offset] -= alpha * delta[offset];
	}
}

void update_output(VALTUE_TYPE *X, VALTUE_TYPE *Y, VALTUE_TYPE *deltaX, VALTUE_TYPE *deltaY, int size, float alpha){
	int num_grid = size / WORK_SIZE + 1;
	dim3 grid(num_grid);
	dim3 block(WORK_SIZE);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	kernel_vector_add <<< grid, block >>>(X, deltaX, size, alpha);
	kernel_vector_add <<< grid, block >>>(Y, deltaY, size, alpha);
}



/*
 * tanimoto distance
 */


__global__ void kernel_tanimoto_distance(VALTUE_TYPE *d_weight_sum, VALTUE_TYPE* d_dist_matrix, size_t n_vec){

	unsigned int row = blockIdx.x;
	unsigned int tid = threadIdx.x;
	unsigned int base = row * n_vec;

	float sum1 = d_weight_sum[row];

	int n = (n_vec + blockDim.x - 1) / blockDim.x;

	for(int i=0; i<n; ++i){
		unsigned int offset = tid + i * blockDim.x;
		if(offset < n_vec){
			float sum2 = d_weight_sum[offset];
			float comm = d_dist_matrix[base + offset];
			float dist = 1 - comm / (sum1 + sum2 - comm);

			if(dist < 0.001){
				dist = 0.01;
			}
			d_dist_matrix[base + offset] = dist;
		}

	}
}



void tanimoto_distance(VALTUE_TYPE *d_data_matrix, VALTUE_TYPE *d_weight_sum, VALTUE_TYPE* d_dist_matrix, size_t n_vec, int dimension){

	int thread_num_per_block = WORK_SIZE;
	while(thread_num_per_block > n_vec){
		thread_num_per_block /= 2;
	}

	dim3 grid(n_vec);
	dim3 block(thread_num_per_block);

	float alpha = 1.0;
	float beta = 0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n_vec, n_vec, dimension, &alpha, d_data_matrix, dimension, d_data_matrix, dimension, &beta, d_dist_matrix, n_vec);


	kernel_tanimoto_distance <<< grid, block>>>(d_weight_sum, d_dist_matrix, n_vec);
}


/*
 * cat gradient from point start to point end
 */
__global__ void kernel_calculate_gradient(VALTUE_TYPE *d_tanimoto_dist, VALTUE_TYPE *d_x_arr, VALTUE_TYPE * d_y_arr,
		VALTUE_TYPE *d_gradient_x, VALTUE_TYPE *d_gradient_y,
		size_t start, size_t end, size_t n_vec){

	unsigned int tid = threadIdx.x;
	unsigned int vec_i = blockIdx.x; // row,
	unsigned int base = vec_i * n_vec;

	unsigned int n = (n_vec + blockDim.x - 1) / blockDim.x;
//	printf("%d\n", tid);

//		for(int vec_j=tid; vec_j<n_vec; vec_j += blockDim.x){
		for(int i=0; i<n; ++i){

			unsigned int vec_j = tid + i * blockDim.x;
			if(vec_j >= n_vec){
				continue;
			}
			int offset = base + vec_j;

			if(vec_i == vec_j){
				d_gradient_x[offset] = 0;
				d_gradient_y[offset] = 0;
					continue;
			}

			float x1 = d_x_arr[vec_i];
			float y1 = d_y_arr[vec_i];

			float x2 = d_x_arr[vec_j];
			float y2 = d_y_arr[vec_j];

			float dx = x1 - x2;
			float dy = y1 - y2;

			float dist_2D = sqrt(pow(dx, 2) + pow(dy, 2));


			float tanimoto_distance = d_tanimoto_dist[offset];

			float c = (dist_2D - tanimoto_distance)/(dist_2D*tanimoto_distance);
//			printf("%f %d\n", tanimoto_distance, offset);

			d_gradient_x[offset] = c * dx;
			d_gradient_y[offset] = c * dy;
//			printf("(%d, %d) %f %f\n", vec_i, vec_j, c*dx, c*dy);


//			printf("%f\n", dx);

		}

}




void caculate_gradient(VALTUE_TYPE *d_tanimoto_dist, VALTUE_TYPE *d_x_arr, VALTUE_TYPE * d_y_arr,
		VALTUE_TYPE *d_gradient_x, VALTUE_TYPE *d_gradient_y,
		size_t start, size_t end, size_t n_vec){

	dim3 grid(end-start);
	dim3 block(WORK_SIZE);

	kernel_calculate_gradient<<< grid, block >>>(d_tanimoto_dist, d_x_arr, d_y_arr,d_gradient_x, d_gradient_y, start, end, n_vec);
}

__global__ void kernel_calculate_gradient2(VALTUE_TYPE *d_gradient_x, VALTUE_TYPE *d_gradient_y,VALTUE_TYPE *d_x_arr, VALTUE_TYPE *d_y_arr, VALTUE_TYPE *d_tanimoto_dist, int n_vec)
{
		unsigned int tid = threadIdx.x;
		unsigned int vec_i = blockIdx.x; // row,
		unsigned int base = vec_i * n_vec;

		__shared__ float sum_x[WORK_SIZE];
		__shared__ float sum_y[WORK_SIZE];

		unsigned int n = (n_vec + blockDim.x - 1) / blockDim.x;
	//	printf("%d\n", tid);


			sum_x[tid] = 0;
			sum_y[tid] = 0;
	//		for(int vec_j=tid; vec_j<n_vec; vec_j += blockDim.x){
			for(int i=0; i<n; ++i){

				unsigned int vec_j = tid + i * blockDim.x;
				if(vec_j >= n_vec){
					continue;
				}
				int offset = base + vec_j;

				if(vec_i == vec_j){
//					d_gradient_x[offset] = 0;
//					d_gradient_y[offset] = 0;
						continue;
				}

				float x1 = d_x_arr[vec_i];
				float y1 = d_y_arr[vec_i];

				float x2 = d_x_arr[vec_j];
				float y2 = d_y_arr[vec_j];

				float dx = x1 - x2;
				float dy = y1 - y2;

				float dist_2D = sqrt(pow(dx, 2) + pow(dy, 2));


				float tanimoto_distance = d_tanimoto_dist[offset];

				float c = (dist_2D - tanimoto_distance)/(dist_2D*tanimoto_distance);
	//			printf("%f %d\n", tanimoto_distance, offset);

//				d_gradient_x[offset] = c * dx;
//				d_gradient_y[offset] = c * dy;
				float a = c * dx;

				sum_x[tid] += a;

				a = c * dy;
				sum_y[tid] += a;
//				printf("(%d, %d) %f %f\n", vec_i, vec_j, c*dx, c*dy);
	//			printf("%f\n", dx);

			}

			__syncthreads();

			for(int half = blockDim.x/2; half != 0; half /= 2){
					if(tid < half){
						sum_x[tid] += sum_x[tid+half];
						sum_y[tid] += sum_y[tid+half];
					}
					__syncthreads();
				}

				if(tid == 0){
					d_gradient_x[vec_i] = sum_x[0];
					d_gradient_y[vec_i] = sum_y[0];
//					printf("%d %f\n", vec_i, sum_x[0]);
			//				printf("%f\n", sum[0]);
				}

}

void caculate_gradient(VALTUE_TYPE *d_gradient_x, VALTUE_TYPE *d_gradient_y, VALTUE_TYPE *d_x_arr, VALTUE_TYPE *d_y_arr, VALTUE_TYPE *d_tanimoto_dist, int n_vec){
	dim3 grid(n_vec);
	dim3 block(WORK_SIZE);

	kernel_calculate_gradient2 <<< grid, block >>>(d_gradient_x, d_gradient_y, d_x_arr, d_y_arr, d_tanimoto_dist, n_vec);
}

/*
 * add by row
 */

__global__ void kernel_row_acc(VALTUE_TYPE* d_arr, VALTUE_TYPE* d_sum, int dimension, int row_offset){
	unsigned int tid = threadIdx.x;
	unsigned int step = blockDim.x;
	unsigned int base =  blockIdx.x * dimension;
	int n = (dimension + blockDim.x - 1) / step;

//	for(int i=0; i<n; ++i){
//		unsigned int offset = tid + i * step;
//		if(offset < dimension){
//			float sum = 0;
//			for(int j=0; j<dimension; ++j){
//				sum += d_arr[offset + j*dimension];
//			}
//			d_arr[offset] = sum;
//		}
//	}


	__shared__ float sum[WORK_SIZE];

	sum[tid] = 0;
	for(int i=0; i<n; i += 1){
		int offset = tid + i * step;
		if(offset < dimension){
			sum[tid] += fabsf(d_arr[base + offset]);
		}

	}
	__syncthreads();

	for(int half = step/2; half != 0; half /= 2){
		if(tid < half){
			sum[tid] += sum[tid+half];
		}
		__syncthreads();
	}

	if(tid == 0){
		d_sum[row_offset+blockIdx.x] = sum[0];
		printf("%d %f\n", blockIdx.x, sum[0]);
//				printf("%f\n", sum[0]);
	}
}

void row_acc(VALTUE_TYPE* d_arr, VALTUE_TYPE* d_sum, int row, int dimension, int row_offset){

	int thread_num_per_block = WORK_SIZE;
	while(thread_num_per_block > dimension){
		thread_num_per_block /= 2;
	}

	dim3 grid(row);
	dim3 block(thread_num_per_block);

	kernel_row_acc<<< grid, block >>>(d_arr, d_sum, dimension, row_offset);
}


/*
 * add by row
 */

__global__ void kernel_row_square_acc(VALTUE_TYPE* d_arr, VALTUE_TYPE* d_sum, int dimension, int row_offset){
	unsigned int tid = threadIdx.x;
	unsigned int step = blockDim.x;
	unsigned int base =  blockIdx.x * dimension;
	int n = (dimension + blockDim.x - 1) / step;


	__shared__ float sum[WORK_SIZE];

	sum[tid] = 0;
	for(int i=0; i<n; i += 1){
		int offset = tid + i * step;
		if(offset < dimension){
			float v = d_arr[base + offset];
			sum[tid] += v * v;
		}

	}
	__syncthreads();

	for(int half = step/2; half != 0; half /= 2){
		if(tid < half){
			sum[tid] += sum[tid+half];
		}
		__syncthreads();
	}

	if(tid == 0){
		d_sum[row_offset+blockIdx.x] = sum[0];
//				printf("%f\n", sum[0]);
	}
}

void row_square_acc(VALTUE_TYPE* d_arr, VALTUE_TYPE* d_sum, int row, int dimension, int row_offset){

	int thread_num_per_block = WORK_SIZE;
	while(thread_num_per_block > dimension){
		thread_num_per_block /= 2;
	}

	dim3 grid(row);
	dim3 block(thread_num_per_block);

	kernel_row_square_acc<<< grid, block >>>(d_arr, d_sum, dimension, row_offset);
}
